#include "hip/hip_runtime.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <cstring>

// CUDA error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            mexPrintf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                     hipGetErrorString(err)); \
            mexErrMsgTxt("CUDA error"); \
        } \
    } while (0)

// CUDA kernel: Compute stress field
__global__ void compute_stress_kernel(
    double *vx, double *vy, double *sigmaxx, double *sigmayy, double *sigmaxy,
    double *memory_dvx_dx, double *memory_dvy_dy,
    double *c11, double *c13, double *c33,
    double *b_x_half, double *b_y, double *a_x_half, double *a_y,
    double *K_x_half, double *K_y,
    double DELTAX, double DELTAY, double DELTAT,
    int NX, int NY)
{
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < NX-1 && j < NY) {
        double value_dvx_dx = (vx[(i + 1) + j * NX] - vx[i + j * NX]) / DELTAX;
        double value_dvy_dy = (vy[i + j * NX] - vy[i + (j - 1) * NX]) / DELTAY;

        memory_dvx_dx[i + j * NX] = b_x_half[i] * memory_dvx_dx[i + j * NX] + 
                                   a_x_half[i] * value_dvx_dx;
        memory_dvy_dy[i + j * NX] = b_y[j] * memory_dvy_dy[i + j * NX] + 
                                   a_y[j] * value_dvy_dy;

        value_dvx_dx = value_dvx_dx / K_x_half[i] + memory_dvx_dx[i + j * NX];
        value_dvy_dy = value_dvy_dy / K_y[j] + memory_dvy_dy[i + j * NX];

        sigmaxx[i + j * NX] += DELTAT * (
            c11[i + j * NX] * value_dvx_dx + 
            c13[i + j * NX] * value_dvy_dy
        );
        
        sigmayy[i + j * NX] += DELTAT * (
            c13[i + j * NX] * value_dvx_dx + 
            c33[i + j * NX] * value_dvy_dy
        );
    }
}

// CUDA kernel: Compute shear stress
__global__ void compute_shear_stress_kernel(
    double *vx, double *vy, double *sigmaxy,
    double *memory_dvy_dx, double *memory_dvx_dy,
    double *c44,
    double *b_x, double *b_y_half, double *a_x, double *a_y_half,
    double *K_x, double *K_y_half,
    double DELTAX, double DELTAY, double DELTAT,
    int NX, int NY)
{
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    if (i < NX && j < NY-1) {
        double value_dvy_dx = (vy[i + j * NX] - vy[(i - 1) + j * NX]) / DELTAX;
        double value_dvx_dy = (vx[i + (j + 1) * NX] - vx[i + j * NX]) / DELTAY;

        memory_dvy_dx[i + j * NX] = b_x[i] * memory_dvy_dx[i + j * NX] + 
                                   a_x[i] * value_dvy_dx;
        memory_dvx_dy[i + j * NX] = b_y_half[j] * memory_dvx_dy[i + j * NX] + 
                                   a_y_half[j] * value_dvx_dy;

        value_dvy_dx = value_dvy_dx / K_x[i] + memory_dvy_dx[i + j * NX];
        value_dvx_dy = value_dvx_dy / K_y_half[j] + memory_dvx_dy[i + j * NX];

        sigmaxy[i + j * NX] += c44[i + j * NX] * (value_dvy_dx + value_dvx_dy) * DELTAT;
    }
}

// CUDA kernel: Compute x-direction velocity field
__global__ void compute_velocity_x_kernel(
    double *vx, double *sigmaxx, double *sigmaxy,
    double *memory_dsigmaxx_dx, double *memory_dsigmaxy_dy,
    double *rho,
    double *b_x, double *b_y, double *a_x, double *a_y,
    double *K_x, double *K_y,
    double DELTAX, double DELTAY, double DELTAT,
    int NX, int NY)
{
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    if (i < NX && j < NY) {
        double value_dsigmaxx_dx = (sigmaxx[i + j * NX] - sigmaxx[(i - 1) + j * NX]) / DELTAX;
        double value_dsigmaxy_dy = (sigmaxy[i + j * NX] - sigmaxy[i + (j - 1) * NX]) / DELTAY;

        memory_dsigmaxx_dx[i + j * NX] = b_x[i] * memory_dsigmaxx_dx[i + j * NX] + 
                                        a_x[i] * value_dsigmaxx_dx;
        memory_dsigmaxy_dy[i + j * NX] = b_y[j] * memory_dsigmaxy_dy[i + j * NX] + 
                                        a_y[j] * value_dsigmaxy_dy;

        value_dsigmaxx_dx = value_dsigmaxx_dx / K_x[i] + memory_dsigmaxx_dx[i + j * NX];
        value_dsigmaxy_dy = value_dsigmaxy_dy / K_y[j] + memory_dsigmaxy_dy[i + j * NX];

        vx[i + j * NX] += (value_dsigmaxx_dx + value_dsigmaxy_dy) * DELTAT / rho[i + j * NX];
    }
}

// CUDA kernel: Compute y-direction velocity field
__global__ void compute_velocity_y_kernel(
    double *vy, double *sigmaxy, double *sigmayy,
    double *memory_dsigmaxy_dx, double *memory_dsigmayy_dy,
    double *rho,
    double *b_x_half, double *b_y_half, double *a_x_half, double *a_y_half,
    double *K_x_half, double *K_y_half,
    double DELTAX, double DELTAY, double DELTAT,
    int NX, int NY)
{
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < NX-1 && j < NY-1) {
        double value_dsigmaxy_dx = (sigmaxy[(i + 1) + j * NX] - sigmaxy[i + j * NX]) / DELTAX;
        double value_dsigmayy_dy = (sigmayy[i + (j + 1) * NX] - sigmayy[i + j * NX]) / DELTAY;

        memory_dsigmaxy_dx[i + j * NX] = b_x_half[i] * memory_dsigmaxy_dx[i + j * NX] + 
                                        a_x_half[i] * value_dsigmaxy_dx;
        memory_dsigmayy_dy[i + j * NX] = b_y_half[j] * memory_dsigmayy_dy[i + j * NX] + 
                                        a_y_half[j] * value_dsigmayy_dy;

        value_dsigmaxy_dx = value_dsigmaxy_dx / K_x_half[i] + memory_dsigmaxy_dx[i + j * NX];
        value_dsigmayy_dy = value_dsigmayy_dy / K_y_half[j] + memory_dsigmayy_dy[i + j * NX];

        vy[i + j * NX] += (value_dsigmaxy_dx + value_dsigmayy_dy) * DELTAT / rho[i + j * NX];
    }
}

// MEX entry function
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    // Check number of input/output parameters
    if (nrhs != 33) {
        mexErrMsgTxt("Need 33 input parameters");
    }
    if (nlhs != 2) {
        mexErrMsgTxt("Need 2 output parameters");
    }

    // Get grid dimensions
    int NX = (int)mxGetScalar(prhs[31]);
    int NY = (int)mxGetScalar(prhs[32]);

    // Get time and space steps
    double DELTAX = mxGetScalar(prhs[28]);
    double DELTAY = mxGetScalar(prhs[29]);
    double DELTAT = mxGetScalar(prhs[30]);

    // Get input array pointers
    double *vx = mxGetPr(prhs[0]);
    double *vy = mxGetPr(prhs[1]);
    double *sigmaxx = mxGetPr(prhs[2]);
    double *sigmayy = mxGetPr(prhs[3]);
    double *sigmaxy = mxGetPr(prhs[4]);
    
    // Get memory variable pointers
    double *memory_dvx_dx = mxGetPr(prhs[5]);
    double *memory_dvy_dy = mxGetPr(prhs[6]);
    double *memory_dvy_dx = mxGetPr(prhs[7]);
    double *memory_dvx_dy = mxGetPr(prhs[8]);
    double *memory_dsigmaxx_dx = mxGetPr(prhs[9]);
    double *memory_dsigmaxy_dy = mxGetPr(prhs[10]);
    double *memory_dsigmaxy_dx = mxGetPr(prhs[11]);
    double *memory_dsigmayy_dy = mxGetPr(prhs[12]);

    // Get material parameter pointers
    double *c11 = mxGetPr(prhs[13]);
    double *c13 = mxGetPr(prhs[14]);
    double *c33 = mxGetPr(prhs[15]);
    double *c44 = mxGetPr(prhs[16]);
    double *rho = mxGetPr(prhs[17]);

    // Get PML parameter pointers
    double *b_x = mxGetPr(prhs[18]);
    double *b_y = mxGetPr(prhs[19]);
    double *b_x_half = mxGetPr(prhs[20]);
    double *b_y_half = mxGetPr(prhs[21]);
    double *a_x = mxGetPr(prhs[22]);
    double *a_y = mxGetPr(prhs[23]);
    double *a_x_half = mxGetPr(prhs[24]);
    double *a_y_half = mxGetPr(prhs[25]);
    double *K_x = mxGetPr(prhs[26]);
    double *K_y = mxGetPr(prhs[27]);
    double *K_x_half = mxGetPr(prhs[28]);
    double *K_y_half = mxGetPr(prhs[29]);

    // Allocate GPU memory - 2D arrays
    size_t size_2d = NX * NY * sizeof(double);
    double *d_vx, *d_vy, *d_sigmaxx, *d_sigmayy, *d_sigmaxy;
    CHECK_CUDA_ERROR(hipMalloc(&d_vx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_vy, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_sigmaxx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_sigmayy, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_sigmaxy, size_2d));

    // Allocate memory for field variables
    double *d_memory_dvx_dx, *d_memory_dvy_dy, *d_memory_dvy_dx, *d_memory_dvx_dy;
    double *d_memory_dsigmaxx_dx, *d_memory_dsigmaxy_dy, *d_memory_dsigmaxy_dx, *d_memory_dsigmayy_dy;
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dvx_dx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dvy_dy, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dvy_dx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dvx_dy, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dsigmaxx_dx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dsigmaxy_dy, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dsigmaxy_dx, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_memory_dsigmayy_dy, size_2d));

    // Copy data to GPU - 2D arrays
    CHECK_CUDA_ERROR(hipMemcpy(d_vx, vx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_vy, vy, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_sigmaxx, sigmaxx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_sigmayy, sigmayy, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_sigmaxy, sigmaxy, size_2d, hipMemcpyHostToDevice));

    // Copy memory variables
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dvx_dx, memory_dvx_dx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dvy_dy, memory_dvy_dy, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dvy_dx, memory_dvy_dx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dvx_dy, memory_dvx_dy, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dsigmaxx_dx, memory_dsigmaxx_dx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dsigmaxy_dy, memory_dsigmaxy_dy, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dsigmaxy_dx, memory_dsigmaxy_dx, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_memory_dsigmayy_dy, memory_dsigmayy_dy, size_2d, hipMemcpyHostToDevice));

    // Allocate memory for material parameters
    double *d_c11, *d_c13, *d_c33, *d_c44, *d_rho;
    CHECK_CUDA_ERROR(hipMalloc(&d_c11, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_c13, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_c33, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_c44, size_2d));
    CHECK_CUDA_ERROR(hipMalloc(&d_rho, size_2d));

    // Copy data to GPU - Material parameters
    CHECK_CUDA_ERROR(hipMemcpy(d_c11, c11, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_c13, c13, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_c33, c33, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_c44, c44, size_2d, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_rho, rho, size_2d, hipMemcpyHostToDevice));

    // Allocate PML parameter pointers - 1D arrays
    size_t size_1d_x = NX * sizeof(double);
    size_t size_1d_y = NY * sizeof(double);
    double *d_b_x, *d_b_y, *d_b_x_half, *d_b_y_half;
    double *d_a_x, *d_a_y, *d_a_x_half, *d_a_y_half;
    double *d_K_x, *d_K_y, *d_K_x_half, *d_K_y_half;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_b_x, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_b_y, size_1d_y));
    CHECK_CUDA_ERROR(hipMalloc(&d_b_x_half, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_b_y_half, size_1d_y));
    CHECK_CUDA_ERROR(hipMalloc(&d_a_x, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_a_y, size_1d_y));
    CHECK_CUDA_ERROR(hipMalloc(&d_a_x_half, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_a_y_half, size_1d_y));
    CHECK_CUDA_ERROR(hipMalloc(&d_K_x, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_K_y, size_1d_y));
    CHECK_CUDA_ERROR(hipMalloc(&d_K_x_half, size_1d_x));
    CHECK_CUDA_ERROR(hipMalloc(&d_K_y_half, size_1d_y));

    // Copy PML parameters - 1D arrays
    CHECK_CUDA_ERROR(hipMemcpy(d_b_x, b_x, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b_y, b_y, size_1d_y, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b_x_half, b_x_half, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b_y_half, b_y_half, size_1d_y, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_a_x, a_x, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_a_y, a_y, size_1d_y, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_a_x_half, a_x_half, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_a_y_half, a_y_half, size_1d_y, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_K_x, K_x, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_K_y, K_y, size_1d_y, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_K_x_half, K_x_half, size_1d_x, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_K_y_half, K_y_half, size_1d_y, hipMemcpyHostToDevice));

    // Set thread block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((NX + blockSize.x - 1) / blockSize.x, 
                  (NY + blockSize.y - 1) / blockSize.y);
    
    // Launch kernels
    compute_stress_kernel<<<gridSize, blockSize>>>(d_vx, d_vy, d_sigmaxx, d_sigmayy, d_sigmaxy,
                                                d_memory_dvx_dx, d_memory_dvy_dy,
                                                d_c11, d_c13, d_c33,
                                                d_b_x_half, d_b_y, d_a_x_half, d_a_y,
                                                d_K_x_half, d_K_y,
                                                DELTAX, DELTAY, DELTAT, NX, NY);

    compute_shear_stress_kernel<<<gridSize, blockSize>>>(d_vx, d_vy, d_sigmaxy,
                                                        d_memory_dvy_dx, d_memory_dvx_dy,
                                                        d_c44,
                                                        d_b_x, d_b_y_half, d_a_x, d_a_y_half,
                                                        d_K_x, d_K_y_half,
                                                        DELTAX, DELTAY, DELTAT, NX, NY);

    compute_velocity_x_kernel<<<gridSize, blockSize>>>(d_vx, d_sigmaxx, d_sigmaxy,
                                                      d_memory_dsigmaxx_dx, d_memory_dsigmaxy_dy,
                                                      d_rho,
                                                      d_b_x, d_b_y, d_a_x, d_a_y,
                                                      d_K_x, d_K_y,
                                                      DELTAX, DELTAY, DELTAT, NX, NY);

    compute_velocity_y_kernel<<<gridSize, blockSize>>>(d_vy, d_sigmaxy, d_sigmayy,
                                                      d_memory_dsigmaxy_dx, d_memory_dsigmayy_dy,
                                                      d_rho,
                                                      d_b_x_half, d_b_y_half, d_a_x_half, d_a_y_half,
                                                      d_K_x_half, d_K_y_half,
                                                      DELTAX, DELTAY, DELTAT, NX, NY);

    // Check for kernel execution errors
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Copy results back to host
    CHECK_CUDA_ERROR(hipMemcpy(vx, d_vx, size_2d, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(vy, d_vy, size_2d, hipMemcpyDeviceToHost));

    // Free GPU memory
    hipFree(d_vx); hipFree(d_vy);
    hipFree(d_sigmaxx); hipFree(d_sigmayy); hipFree(d_sigmaxy);
    hipFree(d_memory_dvx_dx); hipFree(d_memory_dvy_dy);
    hipFree(d_memory_dvy_dx); hipFree(d_memory_dvx_dy);
    hipFree(d_memory_dsigmaxx_dx); hipFree(d_memory_dsigmaxy_dy);
    hipFree(d_memory_dsigmaxy_dx); hipFree(d_memory_dsigmayy_dy);
    hipFree(d_c11); hipFree(d_c13); hipFree(d_c33); hipFree(d_c44);
    hipFree(d_rho);
    hipFree(d_b_x); hipFree(d_b_y);
    hipFree(d_b_x_half); hipFree(d_b_y_half);
    hipFree(d_a_x); hipFree(d_a_y);
    hipFree(d_a_x_half); hipFree(d_a_y_half);
    hipFree(d_K_x); hipFree(d_K_y);
    hipFree(d_K_x_half); hipFree(d_K_y_half);

    // Create output matrices and copy results
    plhs[0] = mxCreateDoubleMatrix(NX, NY, mxREAL);
    plhs[1] = mxCreateDoubleMatrix(NX, NY, mxREAL);
    memcpy(mxGetPr(plhs[0]), vx, NX * NY * sizeof(double));
    memcpy(mxGetPr(plhs[1]), vy, NX * NY * sizeof(double));
} 